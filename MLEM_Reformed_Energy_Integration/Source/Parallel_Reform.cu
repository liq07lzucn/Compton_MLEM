#include "hip/hip_runtime.h"
// My Header Files
#include "../Headers/Includes.cuh"
#include "../Headers/CPUHeaders.cuh"
#include "../Headers/GraphicsCardHeaders.cuh"


int main() {
  float time_start = clock();

  /////////////// Defining initial parameters and variables ///////////////
  // We start by delcaring the input parameters
  string NAME, OUTPUTMOD;
  float x_start, x_end, y_start, y_end, z_start, z_end, CUTOFF;
  unsigned XDIVI, YDIVI, ZDIVI, ITHREADSPB, IBLOCKS, CTHREADSPB, CBLOCKS, TOTALIT, SAVEEVERY, INTSTEP;
  long unsigned CONES;
  bool CORE_OUT, MATT_OUT;

  // Next we import the input parameters from the setup file
  InputSetup( NAME, OUTPUTMOD, x_start, x_end, y_start, y_end, z_start, z_end, XDIVI, YDIVI, ZDIVI,
              ITHREADSPB, IBLOCKS, CONES, CTHREADSPB, CBLOCKS, TOTALIT, SAVEEVERY, INTSTEP, CUTOFF,
              CORE_OUT, MATT_OUT );

  // Now we must check that the input parameters are acceptible
  if ( InputDebugger( XDIVI, YDIVI, ZDIVI, ITHREADSPB, IBLOCKS, CONES, CTHREADSPB, CBLOCKS, INTSTEP, CORE_OUT, MATT_OUT ) != 0 ) return 0;

  // Now we can define the widths of the voxel
  const float delx = (x_end - x_start)/XDIVI;
  const float dely = (y_end - y_start)/YDIVI;
  const float delz = (z_end - z_start)/ZDIVI;

  // This contains the locations of the centres of each voxel
  vector<vector<vector<vector<float> > > > position_matrix(XDIVI, vector<vector<vector<float> > >( YDIVI, vector<vector<float> >( ZDIVI , {0,0,0} ) ) );

  // A flattened list of all the cones, with 10 elements per cone, a single cone has elements:
  // {location of first colision x ,y ,z} , {axis of the cone x ,y ,z} , angle , uncertainty , kn-coefficient
  float* conelist_1D = new float [ CONES * 11 ]{};

  // The f distribution which we want to calculate
  float* f = new float [XDIVI*YDIVI*ZDIVI]{};

  DefinePositions( position_matrix, f, XDIVI, YDIVI, ZDIVI, x_start, y_start, z_start, delx, dely, delz);

  string input = "../Input/Filtered/" + NAME + ".csv";
  CreateCones( conelist_1D, input, CONES);
  hipDeviceReset();




  /////////////// Creating the interaction matrix ///////////////
  cout << "Creating Interaction Matrix:" << '\n';
  float time_section_start = clock();

  unsigned char *voxel_cone_interaction_d; // This flattened matrix is a refference for all the voxels to all the cones.
  hipMalloc( &voxel_cone_interaction_d, XDIVI * YDIVI * ZDIVI * CONES * sizeof(unsigned char) );
  hipMemset( voxel_cone_interaction_d, 0, XDIVI * YDIVI * ZDIVI * CONES * sizeof(unsigned char) );

  float *conelist_1D_d;
  hipMalloc( &conelist_1D_d, CONES * 11 * sizeof(float) );
  hipMemcpy( conelist_1D_d, conelist_1D, CONES * 11 * sizeof(float), hipMemcpyHostToDevice );

  if ( MemDebugger( XDIVI, YDIVI, ZDIVI, CONES ) != 0 ) return 0; // We chack the device memory

  Find_Intersecting<<< CBLOCKS, CTHREADSPB >>>( conelist_1D_d, voxel_cone_interaction_d, XDIVI, YDIVI, ZDIVI, CONES, delx, dely, delz, x_start, y_start, z_start, INTSTEP );
  hipDeviceSynchronize();
  hipFree(conelist_1D_d);

  Print_Time_Complete( time_section_start , clock() );




  /////////////// Now the Iteration - starting with data transfer to the GPU ///////////////
  cout << "Proceeding with Iteration:" << '\n';
  time_section_start = clock();

  float *f_d; // This is the radioactive distribution of the volume, it is what is iterated and what we want to esitmate
  hipMalloc( &f_d , XDIVI * YDIVI * ZDIVI * sizeof(float) );
  hipMemcpy( f_d , f , XDIVI * YDIVI * ZDIVI * sizeof(float) , hipMemcpyHostToDevice );

  float *lambda_vector_d; // This is the vector which will contain the interior sum of the MLEM for each cone
  hipMalloc( &lambda_vector_d, CONES * sizeof(float) );
  hipMemset( lambda_vector_d, 0, CONES * sizeof(float) );

  float *voxel_max_d; // This is the vector which will contain the interior sum of the MLEM for each cone
  hipMalloc( &voxel_max_d, sizeof(float) );
  hipMemset( voxel_max_d, 0, sizeof(float) );

  // Now the iteration process itself
  float time_iter;
  for (unsigned It = 1; It <= TOTALIT; It++) {
    time_iter = clock();

    // GPU Functions are called with the parameters << blocks , threads_per_block >>
    Interior_Sum      <<< CBLOCKS, CTHREADSPB >>> ( f_d, voxel_cone_interaction_d, lambda_vector_d, XDIVI, YDIVI, ZDIVI, CONES );
    Iterate           <<< IBLOCKS, ITHREADSPB >>> ( f_d, voxel_cone_interaction_d, lambda_vector_d, CONES );
    Find_Max          <<< 1, ITHREADSPB, ITHREADSPB*sizeof(float) >>> ( f_d, voxel_max_d, XDIVI, YDIVI, ZDIVI, ITHREADSPB ); // This function has the extra parameter. The amount of shared memory.
    Cull              <<< IBLOCKS, ITHREADSPB >>> ( f_d, voxel_cone_interaction_d, voxel_max_d, CONES, CUTOFF );
    hipDeviceSynchronize();


    // For printing the remaining time of the iterations
    Print_Time_Remaining( time_iter, clock(), It, TOTALIT );

    // Now we copy back and save the f values at the regular intervals
    if (It%SAVEEVERY == 0 || It == 1) {
      hipMemcpy(f, f_d, XDIVI*YDIVI*ZDIVI*sizeof(float), hipMemcpyDeviceToHost);
      string output =  "../Output/" + NAME + OUTPUTMOD + "_C" + std::to_string(CONES) + "_x" + std::to_string(XDIVI) + "y" + std::to_string(YDIVI) + "z" + std::to_string(ZDIVI) + "_I"; // Output file name

      if (MATT_OUT) StoreF_MATT( f, It, output, position_matrix, XDIVI, YDIVI, ZDIVI );
      if (CORE_OUT) StoreF_CORE( f, It, output, XDIVI, YDIVI, ZDIVI, x_start, y_start, z_start, delx, dely, delz );

    }

  }

  Print_Time_Complete( time_section_start, clock() );

  // Now we can free the memory on the GPU and CPU completely
  hipFree(f_d);
  hipFree(voxel_cone_interaction_d);
  hipFree(lambda_vector_d);
  hipFree(voxel_max_d);
  delete [] f;
  delete [] conelist_1D;

  Print_Time_Complete( time_start, clock(), 1 );

  hipDeviceReset();
  return 0;
}








//
